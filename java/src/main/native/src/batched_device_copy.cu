#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "batched_device_copy.hpp"

namespace {

__global__ void batched_copy(cudf::jni::buffer_copy_descr const* descrs) {
  int buffer_idx = blockIdx.x;
  auto dest = static_cast<std::uint8_t*>(descrs[buffer_idx].dest_addr);
  auto src = static_cast<std::uint8_t const*>(descrs[buffer_idx].src_addr);
  auto size = descrs[buffer_idx].size;
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    dest[i] = src[i];
  }
}

} // anonymous namespace

namespace cudf {
namespace jni {

hipError_t batched_memcpy_async(rmm::device_uvector<buffer_copy_descr> const &descrs,
                                 hipStream_t stream) {
  dim3 const grid(descrs.size());

  // copying entire buffer per thread block, so maximize
  // the number of threads per block
  constexpr int block_size = 1024;
  dim3 const block(block_size);

  batched_copy<<<grid, block, 0, stream>>>(descrs.data());
  return hipGetLastError();
}

} // namespace jni
} // namespace cudf
